#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <cstdio>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;


const unsigned int FILTER_SIZE = 25;

__global__ void createFilterImage(unsigned char * inputImage, unsigned char* smoothImage,
    float * filter, const int width, const int height, const int spectrum){
  
  __shared__ float shared_filter[FILTER_SIZE];
  if(threadIdx.x < FILTER_SIZE){
    shared_filter[threadIdx.x] = filter[threadIdx.x];
  }
  __syncthreads();

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(x < width && y < height){

    for ( int z = 0; z < spectrum; z++){
      unsigned int filterItem = 0;
      float filterSum = 0.0f;
      float smoothPix = 0.0f;

      for ( int fy = y - 2; fy < y + 3; fy++ ) {
        if ( fy < 0 ) {
          filterItem += 5;
          continue;
        }
        else if ( fy == height ) {
          break;
        }
        
        for ( int fx = x - 2; fx < x + 3; fx++ ) {
          if ( (fx < 0) || (fx >= width) ) {
            filterItem++;
            continue;
          }

          smoothPix += static_cast< float >(inputImage[(z * width * height) + (fy * width) + fx]) * shared_filter[filterItem];
          filterSum += shared_filter[filterItem];
          filterItem++;
        }
      }
      smoothPix /= filterSum;
      smoothImage[(z * width * height) + (y * width) + x] = static_cast< unsigned char >(smoothPix + 0.5f);
    }
  }
}






float filter[] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 1.0f, 2.0f, 3.0f, 2.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f};

void triangularSmooth(const int width, const int height, const int spectrum, unsigned char * inputImage, unsigned char * smoothImage) {
	NSTimer kernelTime = NSTimer("smooth", false, false);
  hipError_t devRetVal = hipSuccess;
  int img_size = 3 * width * height;


  //Allocate vectors in device memory
  unsigned char * d_input;
  if( (devRetVal = hipMalloc(&d_input, img_size * 
          sizeof(unsigned char))) != hipSuccess){
    cerr << "Impossible to allocate device memory for d_input." << endl;
    cerr << hipGetErrorString(devRetVal) << endl;
    exit(1);
  }
  unsigned char * d_output;
  if( (devRetVal = hipMalloc(&d_output, img_size * sizeof(unsigned char)))
      != hipSuccess){
    cerr << "Impossible to allocate device memory for d_output." << endl;
    exit(1);
  }
  float * d_filter;
  if( (devRetVal = hipMalloc(&d_filter, 25*sizeof(float)))
      != hipSuccess){
    cerr << "Impossible to allocate device memory for d_filter." << endl;
    exit(1);
  }

  //Copy vector from host memory to device memory
  if( (devRetVal = hipMemcpy(d_input, inputImage, img_size * sizeof(unsigned char) , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy inputImage to device" << endl;
    exit(1);
  }
  if( (devRetVal = hipMemcpy(d_output, smoothImage, img_size * sizeof(unsigned char) , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy smoothImage to device" << endl;
    exit(1);
  }
  if( (devRetVal = hipMemcpy(d_filter, filter, 25*sizeof(float) , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy filter to device" << endl;
    exit(1);
  }

  dim3 threadsPerBlock(32,32);
  dim3 numBlocks(width/threadsPerBlock.x, height/threadsPerBlock.y);

  if(width%threadsPerBlock.x != 0){
    numBlocks.x += 1;
  }
  if(height%threadsPerBlock.y != 0){
    numBlocks.y += 1;
  }

  kernelTime.start();
  createFilterImage<<<numBlocks,threadsPerBlock>>>(d_input, d_output, d_filter,
        width, height, spectrum);
  hipDeviceSynchronize();
  kernelTime.stop();

	
  if ( ( devRetVal = hipGetLastError()) != hipSuccess ) {
    cerr << "Kernel has some kind of issue: " << hipGetErrorString(devRetVal)
      << endl;
    exit(1);
  }

  //Copy vector from device memory to host memory
  if ( (devRetVal = hipMemcpy(smoothImage, d_output, (3*height*width) * sizeof(unsigned char), 
          hipMemcpyDeviceToHost)) != hipSuccess){
    cerr << "Impossible to copy d_output to host " << endl;
    exit(1);
  }


  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_filter);


	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << kernelTime.getElapsed() << 
    setprecision(3) << " " << (static_cast< long long unsigned int >(width) 
        * height * 7) / 1000000000.0 / kernelTime.getElapsed() << " " << 
    (static_cast< long long unsigned int >(width) * height * 
     (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() 
    << endl;
}
