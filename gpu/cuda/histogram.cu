#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const unsigned int nrThreads = 1024;
const unsigned int MAX_BLOCKS = 65534;

__global__ void createHistogram(const unsigned char * inputImage, 
    unsigned char * outputImage, unsigned int * histogram, const int width, const int height, int iteration){

  //Convert one-dimensional coordinate to two dimensions
  int x = ((blockIdx.x * blockDim.x) + (threadIdx.x + (iteration * MAX_BLOCKS * nrThreads)))/width;
  int y = ((blockIdx.x * blockDim.x) + (threadIdx.x + (iteration * MAX_BLOCKS * nrThreads)))%width;

  if(x < width && y < height){

    float grayPix = 0.0f;

    float r = static_cast< float >(inputImage[(y * width) + x]);
    float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
    float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

    grayPix = __fadd_rn(__fadd_rn(__fadd_rn(__fmul_rn(0.3f, r),__fmul_rn(0.59f, g)), __fmul_rn(0.11f, b)), 0.5f);

    outputImage[(y*width) + x] = static_cast< unsigned char >(grayPix);
    atomicAdd(&histogram[static_cast< unsigned int >(grayPix)], 1);
  }
}

void histogram1D(const int width, const int height, const unsigned char * inputImage, unsigned char * grayImage, unsigned int * histogram, unsigned char * histogramImage) {
	NSTimer kernelTime = NSTimer("histogram", false, false);
  hipError_t devRetVal = hipSuccess;
  int color_image_size = width*height*3;
  int bw_image_size = width*height;

  
  //Allocate vectors in device memory
  unsigned char * d_input;
  if( (devRetVal = hipMalloc(&d_input, color_image_size * 
          sizeof(unsigned char))) != hipSuccess){
    cerr << "Impossible to allocate device memory for d_input." << endl;
    cerr << hipGetErrorString(devRetVal) << endl;
    exit(1);
  }
  unsigned char * d_output;
  if( (devRetVal = hipMalloc(&d_output, bw_image_size*sizeof(unsigned char)))
      != hipSuccess){
    cerr << "Impossible to allocate device memory for d_output." << endl;
    exit(1);
  }
  unsigned int * d_histogram;
  if( (devRetVal = hipMalloc(&d_histogram, 256*sizeof(unsigned int)))
      != hipSuccess){
    cerr << "Impossible to allocate device memory for d_histogram." << endl;
    exit(1);
  }


  //Copy vector from host memory to device memory
  if( (devRetVal = hipMemcpy(d_input, inputImage, color_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy inputImage to device" << endl;
    exit(1);
  }
  if( (devRetVal = hipMemcpy(d_output, grayImage, bw_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy grayImage to device" << endl;
    exit(1);
  }
  if( (devRetVal = hipMemcpy(d_histogram, histogram, 256*sizeof(unsigned int) , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy histogram to device" << endl;
    exit(1);
  }

  int threadsPerBlock(nrThreads);
  int numBlocks(bw_image_size/nrThreads);

  if(bw_image_size%nrThreads != 0){
    numBlocks++;
  }

	
	kernelTime.start();
	// Kernel
  if(numBlocks > MAX_BLOCKS){
    for(int i = 0;i<=numBlocks/MAX_BLOCKS ; i++){
      createHistogram<<<MAX_BLOCKS, threadsPerBlock>>>(d_input, d_output, d_histogram,
          width,height,i);
      hipDeviceSynchronize();
    }
  }else{
    createHistogram<<<MAX_BLOCKS, threadsPerBlock>>>(d_input, d_output, d_histogram,
        width,height,0);
    hipDeviceSynchronize();
  }

	// /Kernel
	kernelTime.stop();
	
  if ( ( devRetVal = hipGetLastError()) != hipSuccess ) {
    cerr << "Kernel has some kind of issue: " << hipGetErrorString(devRetVal)
      << endl;
    exit(1);
  }

  //Copy vector from device memory to host memory
  if ( (devRetVal = hipMemcpy(grayImage, d_output, bw_image_size, 
          hipMemcpyDeviceToHost)) != hipSuccess){
    cerr << "Impossible to copy d_output to host " << endl;
    exit(1);
  }
  if ( (devRetVal = hipMemcpy(histogram, d_histogram, 256*sizeof(unsigned int), 
          hipMemcpyDeviceToHost)) != hipSuccess){
    cerr << "Impossible to copy d_histogram to host " << endl;
    exit(1);
  }


  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_histogram);


	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << kernelTime.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 6) / 1000000000.0 / kernelTime.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * ((4 * sizeof(unsigned char)) + (1 * sizeof(unsigned int)))) / 1000000000.0 / kernelTime.getElapsed() << endl;
}
