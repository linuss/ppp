#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const unsigned int nrThreads = 1024;
const unsigned int MAX_BLOCKS = 65534;


__global__ void darkenImage(const unsigned char * inputImage,
    unsigned char * outputImage, const int width, const int height, int iteration){

  int x = ((blockIdx.x * blockDim.x) + (threadIdx.x + (iteration * MAX_BLOCKS * nrThreads))) * 3;

  if(x+2 < (3 * width*height)){
    float grayPix = 0.0f;
    float r = static_cast< float >(inputImage[x]);
    float g = static_cast< float >(inputImage[x+1]);
    float b = static_cast< float >(inputImage[x+2]);

    grayPix = __fadd_rn(__fadd_rn(__fmul_rn(0.3f, r),__fmul_rn(0.59f, g)), __fmul_rn(0.11f, b));
    grayPix = fma(grayPix,0.6f,0.5f);


    outputImage[(x/3)] = static_cast< unsigned char >(grayPix);
  }
}




void darkGray(const int width, const int height, 
    const unsigned char * inputImage, unsigned char * darkGrayImage) {
	NSTimer kernelTime = NSTimer("darker", false, false);
  hipError_t devRetVal = hipSuccess;
  int color_image_size = width*height*3;
  int bw_image_size = width*height;

  /*Realign the values in the input image to allow 
    coalesced memory access*/
  unsigned char * inputImageCoalesced = (unsigned char *)
    (malloc(3*height*width*(sizeof(unsigned char))));
  
  for(int i = 0; i<bw_image_size;i++){
    inputImageCoalesced[i*3] = inputImage[i];
    inputImageCoalesced[(i*3)+1] = inputImage[bw_image_size + i];
    inputImageCoalesced[(i*3)+2] = inputImage[(2*bw_image_size) + i];
  }

  //Allocate vectors in device memory
  unsigned char * d_input;
  if( (devRetVal = hipMalloc(&d_input, color_image_size * 
          sizeof(unsigned char))) != hipSuccess){
    cerr << "Impossible to allocate device memory for d_input." << endl;
    cerr << hipGetErrorString(devRetVal) << endl;
    exit(1);
  }
  unsigned char * d_output;
  if( (devRetVal = hipMalloc(&d_output, bw_image_size*sizeof(unsigned char)))
      != hipSuccess){
    cerr << "Impossible to allocate device memory for d_output." << endl;
    exit(1);
  }


  if( (devRetVal = hipMemcpy(d_input, inputImageCoalesced, color_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy inputImage to device" << endl;
    exit(1);
  }

  if( (devRetVal = hipMemcpy(d_output, darkGrayImage, bw_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy darkGrayImage to device" << endl;
    exit(1);
  }


  int threadsPerBlock(nrThreads);
  int numBlocks((bw_image_size/nrThreads) );

  if(bw_image_size%nrThreads != 0){
    numBlocks++;
  }

  free(inputImageCoalesced);

	kernelTime.start();
	if(numBlocks > MAX_BLOCKS){
	    for(int i = 0; i<=numBlocks/MAX_BLOCKS ; i++){

	      darkenImage<<<MAX_BLOCKS, threadsPerBlock>>>(d_input, d_output, width,
		  height,i);
	      hipDeviceSynchronize();
	    }
	}else{
	  darkenImage<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width,height,0);
	  hipDeviceSynchronize();
	  }
  kernelTime.stop();

  if ( ( devRetVal = hipGetLastError()) != hipSuccess ) {
    cerr << "Kernel has some kind of issue: " << hipGetErrorString(devRetVal)
      << endl;
    exit(1);
  }

  //Copy vector from device memory to host memory
  if ( (devRetVal = hipMemcpy(darkGrayImage, d_output, bw_image_size, 
          hipMemcpyDeviceToHost)) != hipSuccess){
    cerr << "Impossible to copy d_output to host " << endl;
    exit(1);
  }

  hipFree(d_input);
  hipFree(d_output);



	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << kernelTime.getElapsed() << 
    setprecision(3) << " " << (static_cast< long long unsigned int >(width) 
        * height * 7) / 1000000000.0 / kernelTime.getElapsed() << " " << 
    (static_cast< long long unsigned int >(width) * height * 
     (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() 
    << endl;
}
