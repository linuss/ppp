#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const unsigned int nrThreads = 256;

__global__ void darkenImage(const unsigned char * inputImage, 
    unsigned char * outputImage, const int width, const int height){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  if(x < width && y < height){

    float grayPix = 0.0f;

    float r = 
      static_cast< float >(inputImage[(y * width) + x]);
    float g = 
      static_cast< float >(inputImage[(width * height) + (y * width) + x]);
    float b = 
      static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

    grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
    grayPix = (grayPix * 0.6f) + 0.5f;

    outputImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
  }
}



void darkGray(const int width, const int height, 
    const unsigned char * inputImage, unsigned char * darkGrayImage) {
	NSTimer kernelTime = NSTimer("darker", false, false);
  hipError_t devRetVal = hipSuccess;
  int color_image_size = width*height*3;
  int bw_image_size = width*height;

  
  //Allocate vectors in device memory
  unsigned char * d_input;
  if( (devRetVal = hipMalloc(&d_input, color_image_size * 
          sizeof(unsigned char))) != hipSuccess){
    cerr << "Impossible to allocate device memory for d_input." << endl;
    cerr << hipGetErrorString(devRetVal) << endl;
    exit(1);
  }
  unsigned char * d_output;
  if( (devRetVal = hipMalloc(&d_output, bw_image_size*sizeof(unsigned char)))
      != hipSuccess){
    cerr << "Impossible to allocate device memory for d_output." << endl;
    exit(1);
  }


  //Copy vector from host memory to device memory
  if( (devRetVal = hipMemcpy(d_input, inputImage, color_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy inputImage to device" << endl;
    exit(1);
  }
  if( (devRetVal = hipMemcpy(d_output, darkGrayImage, bw_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy darkGrayImage to device" << endl;
    exit(1);
  }

  dim3 threadsPerBlock(32,32);
  dim3 numBlocks(color_image_size/threadsPerBlock.x, color_image_size/threadsPerBlock.y);

	kernelTime.start();
  darkenImage<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width,
      height);
  hipDeviceSynchronize();
	kernelTime.stop();

  if ( ( devRetVal = hipGetLastError()) != hipSuccess ) {
    cerr << "Kernel has some kind of issue: " << hipGetErrorString(devRetVal)
      << endl;
    exit(1);
  }

  //Copy vector from device memory to host memory
  if ( (devRetVal = hipMemcpy(darkGrayImage, d_output, bw_image_size, 
          hipMemcpyDeviceToHost)) != hipSuccess){
    cerr << "Impossible to copy d_output to host " << endl;
    exit(1);
  }

  hipFree(d_input);
  hipFree(d_output);
	
	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << kernelTime.getElapsed() << 
    setprecision(3) << " " << (static_cast< long long unsigned int >(width) 
        * height * 7) / 1000000000.0 / kernelTime.getElapsed() << " " << 
    (static_cast< long long unsigned int >(width) * height * 
     (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() 
    << endl;
}
