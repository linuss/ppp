#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const unsigned int nrThreads = 1024;
const unsigned int MAX_BLOCKS = 65534;


__global__ void darkenImage(const unsigned char * inputImage,
    unsigned char * outputImage, const int width, const int height, int iteration){

  int x = ((blockIdx.x * blockDim.x) + (threadIdx.x + (iteration * MAX_BLOCKS * nrThreads)))%width;
  int y = ((blockIdx.x * blockDim.x) + (threadIdx.x + (iteration * MAX_BLOCKS * nrThreads)))/width;

  if(x < width && y < height){
    double grayPix = 0.0f;
    double r = static_cast< double >(inputImage[(y * width) + x]);
    double g = static_cast< double >(inputImage[(width * height) + (y * width) + x]);
    double b = static_cast< double >(inputImage[(2 * width * height) + (y * width) + x]);

    grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
    grayPix = (grayPix * 0.6f) + 0.5f;

    outputImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
  }
}




void darkGray(const int width, const int height, 
    const unsigned char * inputImage, unsigned char * darkGrayImage) {
	NSTimer kernelTime = NSTimer("darker", false, false);
  hipError_t devRetVal = hipSuccess;
  int color_image_size = width*height*3;
  int bw_image_size = width*height;

  //Allocate vectors in device memory
  unsigned char * d_input;
  if( (devRetVal = hipMalloc(&d_input, color_image_size * 
          sizeof(unsigned char))) != hipSuccess){
    cerr << "Impossible to allocate device memory for d_input." << endl;
    cerr << hipGetErrorString(devRetVal) << endl;
    exit(1);
  }
  unsigned char * d_output;
  if( (devRetVal = hipMalloc(&d_output, bw_image_size*sizeof(unsigned char)))
      != hipSuccess){
    cerr << "Impossible to allocate device memory for d_output." << endl;
    exit(1);
  }


  //Copy vector from host memory to device memory
  if( (devRetVal = hipMemcpy(d_input, inputImage, color_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy inputImage to device" << endl;
    exit(1);
  }
  if( (devRetVal = hipMemcpy(d_output, darkGrayImage, bw_image_size , 
          hipMemcpyHostToDevice)) != hipSuccess){
    cerr << "Impossible to copy darkGrayImage to device" << endl;
    exit(1);
  }

  int threadsPerBlock(nrThreads);
  int numBlocks((bw_image_size/nrThreads) );

  if(bw_image_size%nrThreads != 0){
    numBlocks++;
  }


	kernelTime.start();
	if(numBlocks > MAX_BLOCKS){
	    for(int i = 0; i<=numBlocks/MAX_BLOCKS ; i++){

	      darkenImage<<<MAX_BLOCKS, threadsPerBlock>>>(d_input, d_output, width,
		  height,i);
	      hipDeviceSynchronize();
	    }
	}else{
	  darkenImage<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width,height,0);
	  hipDeviceSynchronize();
	  }
  kernelTime.stop();

  if ( ( devRetVal = hipGetLastError()) != hipSuccess ) {
    cerr << "Kernel has some kind of issue: " << hipGetErrorString(devRetVal)
      << endl;
    exit(1);
  }

  //Copy vector from device memory to host memory
  if ( (devRetVal = hipMemcpy(darkGrayImage, d_output, bw_image_size, 
          hipMemcpyDeviceToHost)) != hipSuccess){
    cerr << "Impossible to copy d_output to host " << endl;
    exit(1);
  }

  hipFree(d_input);
  hipFree(d_output);

  unsigned char outputImage2[bw_image_size];
	
  for(int x=0;x<width;x++){
    for(int y=0;y<height;y++){
      float grayPix = 0.0f;
      float r = static_cast< float >(inputImage[(y * width) + x]);
      float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
      float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

      grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
      grayPix = (grayPix * 0.6f) + 0.5f;

      outputImage2[(y * width) + x] = static_cast< unsigned char >(grayPix);
    }
  }

  for(int x=0;x<width;x++){
    for(int y=0;y<height;y++){
      if(darkGrayImage[(y*width) + x] != (outputImage2[(y*width) + x])){
        printf("Pixel %d,%d differs - Assigned by thread nr %d\n", x,y,y*width + x );
        printf("Value in darkGrayImage: %d. Value in outputImage2: %d\n", static_cast< unsigned int >(darkGrayImage[(y*width) + x]),static_cast< unsigned int >(outputImage2[(y*width) + x]));
      }
    }
  }
        


	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << kernelTime.getElapsed() << 
    setprecision(3) << " " << (static_cast< long long unsigned int >(width) 
        * height * 7) / 1000000000.0 / kernelTime.getElapsed() << " " << 
    (static_cast< long long unsigned int >(width) * height * 
     (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() 
    << endl;
}
